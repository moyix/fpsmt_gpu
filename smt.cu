#include "hip/hip_runtime.h"
#include <time.h>
#include <sys/time.h>
#include <sched.h>
#include "hiprand/hiprand_kernel.h"
#include "SMTLIB/Float.h"
#include "SMTLIB/BufferRef.h"
#include "cuda_aes.h"
#include "theory.h"
#include "smt.h"

#define AES_BLOCK_SIZE 16

// should come from theory.cu
extern int varsize;

__device__ int solved = 0;
volatile int finished_dev = 0;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  } else {
    // fprintf(stderr,"Success: %s %s %d\n", hipGetErrorString(code), file,
    // line);
  }
}

__host__ __device__ inline int64_t aes_pad(int64_t num) {
  return (num + AES_BLOCK_SIZE - 1) & -AES_BLOCK_SIZE;
}

// Note: size is the *unpadded* size of the input vars
//__global__ void fuzz(uint8_t *in_data, size_t size, const uint8_t *key, uint64_t *gobuf, unsigned long long *execs) {
__global__ void fuzz(uint8_t *in_data, size_t size, hiprandState *state, uint64_t *gobuf, unsigned long long *execs) {
  int bindex = blockIdx.x * blockDim.x + threadIdx.x;

  uint8_t *data = in_data + bindex*size; // i think?

  hiprandState localState = state[bindex];

  while (!solved) {
    atomicAdd(execs, 1);
    // Randomize input for our slice
    uint8_t* curr = data;
    //TODO: once we confirm 16bytes and we generate 8bytes, replace loop with writes
    while (curr < data + size)
    {
        *curr++ = hiprand(&localState); //TODO: i think this is 8bytes but not sure, alternative is uint4
    }

    if (LLVMFuzzerTestOneInput(data, size)) {
      *gobuf = bindex;
      memcpy(in_data+offset, sdata+soff, size);
      solved = 1;
    }
    // Add increment to randomize (I hope?)
    for (int i = 0; i < padded; i += AES_BLOCK_SIZE) {
      *(uint64_t *)(sdata+soff+i) = bindex * (padded/AES_BLOCK_SIZE) + i;
    }
  }
  return;
}

void CUDART_CB finishedCB(void *data) {
  finished_dev = *(int *)data;
}

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int seed = id*37;
    hiprand_init(seed, id, 0, &state[id]);
}

void launch_kernel(int device, int varsize, uint8_t **ret_gbuf, uint64_t **ret_gobuf, unsigned long long **ret_execs) {
  hipSetDevice(device);

  uint8_t *gbuf;
  uint64_t *gobuf;
  unsigned long long *gexecs;

  int size = varsize; // i think?
  hiprandState *rngStates;
  gpuErrchk(hipMalloc(&rngStates, N*M*sizeof(hiprandState)));

  setup_kernel<<<M,N>>>(rngStates);

  // Alloc GPU buffers
  gpuErrchk(hipMalloc(&gbuf, size*N*M));
  gpuErrchk(hipMalloc(&gobuf, sizeof(uint64_t)));
  gpuErrchk(hipMalloc(&gexecs, sizeof(unsigned long long)));

  *ret_gbuf = gbuf;
  *ret_gobuf = gobuf;
  *ret_execs = gexecs;

  // Start fuzzing!
  hipStream_t stream;
  gpuErrchk(hipStreamCreate(&stream));
  int *dev = (int *)malloc(sizeof(int));
  *dev = device + 1;
  printf("Launching kernel on GPU%d...\n", device);
  //fuzz<<<M,N,0,stream>>>(gbuf, varsize, rnd, gobuf, gexecs);
  fuzz<<<M,N,0,stream>>>(gbuf, varsize, rngStates, gobuf, gexecs);
  gpuErrchk(hipLaunchHostFunc(stream, finishedCB, dev));
  //gpuErrchk(hiprandDestroyGenerator(gen));
}

int main(int argc, char **argv) {
  int NUM_GPU;
  gpuErrchk( hipGetDeviceCount(&NUM_GPU) );
  if (NUM_GPU < 1) {
    fprintf(stderr, "No CUDA-capable GPUs detected!\n");
    return 1;
  }

  uint8_t *gbuf[NUM_GPU];
  uint64_t *gobuf[NUM_GPU];
  unsigned long long *goexecs[NUM_GPU];

  struct timespec begin, end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &begin);
  for (int i = 0; i < NUM_GPU; i++) {
    launch_kernel(i, varsize, &gbuf[i], &gobuf[i], &goexecs[i]);
  }

  printf("Waiting on GPUs...\n");
  while (!finished_dev) sched_yield();
  int i = finished_dev - 1;
  // Wait to finish
  hipSetDevice(i);
  hipDeviceSynchronize();
  unsigned long long hexecs;
  printf("Search completed on device %d\n", i);
  gpuErrchk(hipMemcpy(&hexecs, goexecs[i], sizeof(unsigned long long), hipMemcpyDeviceToHost));
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  float seconds = (end.tv_nsec - begin.tv_nsec) / 1000000000.0 + (end.tv_sec  - begin.tv_sec);
  printf("Did %llu execs in %f seconds, %f execs/s\n", hexecs, seconds, hexecs / seconds);


  // Get and print output
  int64_t padded = aes_pad(varsize);
  uint8_t *buf = (uint8_t *)malloc(padded);
  uint64_t oindex;
  gpuErrchk(hipMemcpy(&oindex, gobuf[i], sizeof(uint64_t), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(buf, gbuf[i]+(oindex*padded), padded, hipMemcpyDeviceToHost));
  printf("Found a satisfying assignment on device %d thread %lu:\n", i, oindex);
  for (int k = 0; k < varsize; k++) printf("%02x", buf[k]); printf("\n");
}
