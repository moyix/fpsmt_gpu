#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                                     JFS
//
// Copyright 2017-2018 Daniel Liew
//
// This file is distributed under the MIT license.
// See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
// This is the implemenation of the runtime for SMTLIB BitVectors that
// uses native machine operations. It is written with a C compatible interface
// so that in the future we can easily use LLVM's JIT.

#include "SMTLIB/NativeBitVector.h"
#include "SMTLIB/jassert.h"

// Helper constants/functions
namespace {

const jfs_nr_width_ty jfs_nr_bitvector_ty_bit_width =
    sizeof(jfs_nr_bitvector_ty) * 8;

__device__ jfs_nr_bitvector_ty jfs_nr_get_bitvector_mask(const jfs_nr_width_ty bitWidth) {
  static_assert(jfs_nr_bitvector_ty_bit_width <= 64, "Wrong width");
  jassert(bitWidth <= jfs_nr_bitvector_ty_bit_width);
  return (bitWidth >= jfs_nr_bitvector_ty_bit_width)
             ? UINT64_MAX
             : ((UINT64_C(1) << bitWidth) - 1);
}


__device__ jfs_nr_bitvector_ty
jfs_nr_get_most_signficiant_bit_mask(const jfs_nr_width_ty bitWidth) {
  jassert(bitWidth <= jfs_nr_bitvector_ty_bit_width);
  return (UINT64_C(1) << (bitWidth - 1));
}

}

#ifdef __cplusplus
extern "C" {
#endif

// Public functions

__device__ jfs_nr_bitvector_ty jfs_nr_get_bitvector_mod(const jfs_nr_bitvector_ty value,
                                             const jfs_nr_width_ty bitWidth) {
  static_assert(jfs_nr_bitvector_ty_bit_width <= 64, "Wrong width");
  if (bitWidth >= jfs_nr_bitvector_ty_bit_width) {
    return value;
  } else {
    return value % (UINT64_C(1) << bitWidth);
  }
}

__device__ bool jfs_nr_is_valid(const jfs_nr_bitvector_ty value,
                     const jfs_nr_width_ty width) {
  return jfs_nr_get_bitvector_mod(value, width) == value;
}

__device__ jfs_nr_bitvector_ty jfs_nr_concat(const jfs_nr_bitvector_ty lhs,
                                  const jfs_nr_width_ty lhsBitWidth,
                                  const jfs_nr_bitvector_ty rhs,
                                  const jfs_nr_width_ty rhsBitWidth) {
  jassert(jfs_nr_is_valid(lhs, lhsBitWidth));
  jassert(jfs_nr_is_valid(rhs, rhsBitWidth));
  jassert(((lhsBitWidth + rhsBitWidth) <= jfs_nr_bitvector_ty_bit_width) &&
          "concat too wide");
  jfs_nr_bitvector_ty newValue = rhs;
  newValue |= (lhs << rhsBitWidth);
  return newValue;
}

// Extract bits [highBit, lowBit]
__device__ jfs_nr_bitvector_ty jfs_nr_extract(const jfs_nr_bitvector_ty value,
                                   const jfs_nr_width_ty bitWidth,
                                   const jfs_nr_width_ty highBit,
                                   const jfs_nr_width_ty lowBit) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert(highBit >= lowBit && "Invalid highBit and/or lowBit value");
  jassert(highBit < bitWidth && "Invalid highBit bit");
  jassert(lowBit < bitWidth && "Invalid lowBit bit");
  const jfs_nr_width_ty newWidth = (highBit - lowBit) + 1;
  if (newWidth == bitWidth)
    return value;
  jfs_nr_bitvector_ty temp = value;
  // Remove higher bits that we don't want
  jfs_nr_bitvector_ty mask = jfs_nr_get_bitvector_mask(highBit + 1);
  temp &= mask;

  // Remove lower bits that we don't want.
  temp >>= lowBit;
  jassert(jfs_nr_is_valid(temp, newWidth));
  return temp;
}

// Zero extend to bitvector (bitWidth + extraBits) wide
__device__ jfs_nr_bitvector_ty jfs_nr_zero_extend(const jfs_nr_bitvector_ty value,
                                       const jfs_nr_width_ty bitWidth,
                                       const jfs_nr_width_ty extraBits) {
  // No really work to do provided internal invariant that unused biits
  // are zero is maintained.
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert((bitWidth + extraBits) <= jfs_nr_bitvector_ty_bit_width);
  return value;
}

__device__ jfs_nr_bitvector_ty jfs_nr_sign_extend(const jfs_nr_bitvector_ty value,
                                       const jfs_nr_width_ty bitWidth,
                                       const jfs_nr_width_ty extraBits) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert((bitWidth + extraBits) <= jfs_nr_bitvector_ty_bit_width);
  if (value & jfs_nr_get_most_signficiant_bit_mask(bitWidth)) {
    // msb is not zero. Must do sign extend with ones.
    const jfs_nr_bitvector_ty currentWidthMask =
        jfs_nr_get_bitvector_mask(bitWidth);
    const jfs_nr_bitvector_ty newWidthMask =
        jfs_nr_get_bitvector_mask(bitWidth + extraBits);
    return (value | (~currentWidthMask)) & newWidthMask;
  } else {
    // Just do zero extend
    return jfs_nr_zero_extend(value, bitWidth, extraBits);
  }
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvneg(const jfs_nr_bitvector_ty value,
                                 const jfs_nr_width_ty bitWidth) {
  // [[(bvneg s)]] := nat2bv[m](2^m - bv2nat([[s]]))
  jassert(jfs_nr_is_valid(value, bitWidth));
  if (value == 0) {
    return 0;
  }

  // In two's complement, flipping bits and adding one negates
  // the number.
  return ((~value) & jfs_nr_get_bitvector_mask(bitWidth)) + 1;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvadd(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  // [[(bvadd s t)]] := nat2bv[m](bv2nat([[s]]) + bv2nat([[t]]))
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return jfs_nr_get_bitvector_mod(lhs + rhs, bitWidth);
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvsub(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  // (bvsub s t) abbreviates (bvadd s (bvneg t))
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  // TODO: Verify the implementation is semantically equivalent
  // to SMT-LIBv2
  return jfs_nr_get_bitvector_mod(lhs - rhs, bitWidth);
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvmul(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  // [[(bvmul s t)]] := nat2bv[m](bv2nat([[s]]) * bv2nat([[t]]))
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return jfs_nr_get_bitvector_mod(lhs * rhs, bitWidth);
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvudiv(const jfs_nr_bitvector_ty dividend,
                                  const jfs_nr_bitvector_ty divisor,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(dividend, bitWidth));
  jassert(jfs_nr_is_valid(divisor, bitWidth));
  //   [[(bvudiv s t)]] := if bv2nat([[t]]) = 0
  //                       then λx:[0, m). 1
  //                       else nat2bv[m](bv2nat([[s]]) div bv2nat([[t]]))
  if (divisor == 0) {
    return jfs_nr_get_bitvector_mask(bitWidth);
  }
  jfs_nr_bitvector_ty result = dividend / divisor;
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvurem(const jfs_nr_bitvector_ty dividend,
                                  const jfs_nr_bitvector_ty divisor,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(dividend, bitWidth));
  jassert(jfs_nr_is_valid(divisor, bitWidth));
  //  [[(bvurem s t)]] := if bv2nat([[t]]) = 0
  //                      then [[s]]
  //                      else nat2bv[m](bv2nat([[s]]) rem bv2nat([[t]]))
  jfs_nr_bitvector_ty result = 0;
  if (divisor == 0) {
    result = dividend;
  } else {
    result = dividend % divisor;
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvsdiv(const jfs_nr_bitvector_ty dividend,
                                  const jfs_nr_bitvector_ty divisor,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(dividend, bitWidth));
  jassert(jfs_nr_is_valid(divisor, bitWidth));
  // (bvsdiv s t) abbreviates
  //  (let ((?msb_s ((_ extract |m-1| |m-1|) s))
  //        (?msb_t ((_ extract |m-1| |m-1|) t)))
  //    (ite (and (= ?msb_s #b0) (= ?msb_t #b0))
  //         (bvudiv s t)
  //    (ite (and (= ?msb_s #b1) (= ?msb_t #b0))
  //         (bvneg (bvudiv (bvneg s) t))
  //    (ite (and (= ?msb_s #b0) (= ?msb_t #b1))
  //         (bvneg (bvudiv s (bvneg t)))
  //         (bvudiv (bvneg s) (bvneg t))))))
  const jfs_nr_bitvector_ty msbMask =
      jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  bool msb_s = dividend & msbMask;
  bool msb_t = divisor & msbMask;
  jfs_nr_bitvector_ty result = 0;
  // TODO: Can we write this more efficiently?
  if (!msb_s && !msb_t) {
    // Both operands are postive in two's complement
    result = jfs_nr_bvudiv(dividend, divisor, bitWidth);
  } else if (msb_s && !msb_t) {
    // lhs is negative and rhs is positive in two's complement
    result = jfs_nr_bvneg(
        jfs_nr_bvudiv(jfs_nr_bvneg(dividend, bitWidth), divisor, bitWidth),
        bitWidth);
  } else if (!msb_s && msb_t) {
    // lhs is positive and rhs is negative in two's complement
    result = jfs_nr_bvneg(
        jfs_nr_bvudiv(dividend, jfs_nr_bvneg(divisor, bitWidth), bitWidth),
        bitWidth);
  } else {
    // Both operands are negative in two's complement
    result = jfs_nr_bvudiv(jfs_nr_bvneg(dividend, bitWidth),
                           jfs_nr_bvneg(divisor, bitWidth), bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvsrem(const jfs_nr_bitvector_ty dividend,
                                  const jfs_nr_bitvector_ty divisor,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(dividend, bitWidth));
  jassert(jfs_nr_is_valid(divisor, bitWidth));
  // 2's complement signed remainder (sign follows dividend)
  // (bvsrem s t) abbreviates
  //  (let ((?msb_s ((_ extract |m-1| |m-1|) s))
  //        (?msb_t ((_ extract |m-1| |m-1|) t)))
  //    (ite (and (= ?msb_s #b0) (= ?msb_t #b0))
  //         (bvurem s t)
  //    (ite (and (= ?msb_s #b1) (= ?msb_t #b0))
  //         (bvneg (bvurem (bvneg s) t))
  //    (ite (and (= ?msb_s #b0) (= ?msb_t #b1))
  //         (bvurem s (bvneg t)))
  //         (bvneg (bvurem (bvneg s) (bvneg t))))))
  const jfs_nr_bitvector_ty msbMask =
      jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  bool msb_s = dividend & msbMask;
  bool msb_t = divisor & msbMask;
  jfs_nr_bitvector_ty result = 0;
  // TODO: Can we write this more efficiently?
  if (!msb_s && !msb_t) {
    // Both operands are postive in two's complement
    result = jfs_nr_bvurem(dividend, divisor, bitWidth);
  } else if (msb_s && !msb_t) {
    // lhs is negative and rhs is positive in two's complement
    result = jfs_nr_bvneg(
        jfs_nr_bvurem(jfs_nr_bvneg(dividend, bitWidth), divisor, bitWidth),
        bitWidth);
  } else if (!msb_s && msb_t) {
    // lhs is positive and rhs is negative in two's complement
    result = jfs_nr_bvurem(dividend, jfs_nr_bvneg(divisor, bitWidth), bitWidth);
  } else {
    // Both operands are negative in two's complement
    result =
        jfs_nr_bvneg(jfs_nr_bvurem(jfs_nr_bvneg(dividend, bitWidth),
                                   jfs_nr_bvneg(divisor, bitWidth), bitWidth),
                     bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvsmod(const jfs_nr_bitvector_ty dividend,
                                  const jfs_nr_bitvector_ty divisor,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(dividend, bitWidth));
  jassert(jfs_nr_is_valid(divisor, bitWidth));
  // 2's complement signed remainder (sign follows divisor)
  // (bvsmod s t) abbreviates
  //  (let ((?msb_s ((_ extract |m-1| |m-1|) s))
  //        (?msb_t ((_ extract |m-1| |m-1|) t)))
  //    (let ((abs_s (ite (= ?msb_s #b0) s (bvneg s)))
  //          (abs_t (ite (= ?msb_t #b0) t (bvneg t))))
  //      (let ((u (bvurem abs_s abs_t)))
  //        (ite (= u (_ bv0 m))
  //             u
  //        (ite (and (= ?msb_s #b0) (= ?msb_t #b0))
  //             u
  //        (ite (and (= ?msb_s #b1) (= ?msb_t #b0))
  //             (bvadd (bvneg u) t)
  //        (ite (and (= ?msb_s #b0) (= ?msb_t #b1))
  //             (bvadd u t)
  //             (bvneg u))))))))
  // TODO: Can we write this more efficiently?
  const jfs_nr_bitvector_ty msbMask =
      jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  bool msb_s = dividend & msbMask;
  bool msb_t = divisor & msbMask;
  jfs_nr_bitvector_ty abs_s =
      msb_s ? jfs_nr_bvneg(dividend, bitWidth) : dividend;
  jfs_nr_bitvector_ty abs_t = msb_t ? jfs_nr_bvneg(divisor, bitWidth) : divisor;
  jfs_nr_bitvector_ty u = jfs_nr_bvurem(abs_s, abs_t, bitWidth);
  jfs_nr_bitvector_ty result = 0;
  if (u == 0) {
    result = u;
  } else if (!msb_s && !msb_t) {
    // Both operands are postive in two's complement
    result = u;
  } else if (msb_s && !msb_t) {
    // lhs is negative and rhs is positive in two's complement
    result = jfs_nr_bvadd(jfs_nr_bvneg(u, bitWidth), divisor, bitWidth);
  } else if (!msb_s && msb_t) {
    // lhs is positive and rhs is negative in two's complement
    result = jfs_nr_bvadd(u, divisor, bitWidth);
  } else {
    // Both operands are negative in two's complement
    result = jfs_nr_bvneg(u, bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvshl(const jfs_nr_bitvector_ty value,
                                 const jfs_nr_bitvector_ty shift,
                                 const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert(jfs_nr_is_valid(shift, bitWidth));
  //  [[(bvshl s t)]] := nat2bv[m](bv2nat([[s]]) * 2^(bv2nat([[t]])))
  jfs_nr_bitvector_ty result = 0;
  if (shift >= bitWidth) {
    // Overshift to zero
    result = 0;
  } else {
    result = jfs_nr_get_bitvector_mod(value << shift, bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvlshr(const jfs_nr_bitvector_ty value,
                                  const jfs_nr_bitvector_ty shift,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert(jfs_nr_is_valid(shift, bitWidth));
  // [[(bvlshr s t)]] := nat2bv[m](bv2nat([[s]]) div 2^(bv2nat([[t]])))
  jfs_nr_bitvector_ty result = 0;
  if (shift >= bitWidth) {
    // Overshift to zero
    result = 0;
  } else {
    result = jfs_nr_get_bitvector_mod(value >> shift, bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvashr(const jfs_nr_bitvector_ty value,
                                  const jfs_nr_bitvector_ty shift,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jassert(jfs_nr_is_valid(shift, bitWidth));
  // (bvashr s t) abbreviates
  //  (ite (= ((_ extract |m-1| |m-1|) s) #b0)
  //       (bvlshr s t)
  //       (bvnot (bvlshr (bvnot s) t)))
  // TODO: Can we do this more efficiently?
  bool lhsNeg = value & jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  jfs_nr_bitvector_ty result = 0;
  if (!lhsNeg) {
    result = jfs_nr_bvlshr(value, shift, bitWidth);
  } else {
    result = jfs_nr_bvnot(
        jfs_nr_bvlshr(jfs_nr_bvnot(value, bitWidth), shift, bitWidth),
        bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_rotate_left(const jfs_nr_bitvector_ty value,
                                       const jfs_nr_width_ty shift,
                                       const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  // ((_ rotate_left 0) t) stands for t
  // ((_ rotate_left i) t) abbreviates t if m = 1, and
  //   ((_ rotate_left |i-1|)
  //     (concat ((_ extract |m-2| 0) t) ((_ extract |m-1| |m-1|) t))
  //   otherwise
  jfs_nr_bitvector_ty effective_shift = shift % bitWidth;
  // Shift bits to the left
  jassert(effective_shift < jfs_nr_bitvector_ty_bit_width);
  jfs_nr_bitvector_ty result = value << effective_shift;
  // bitwise or with the bits that got shifted out and
  // should now appear in the lsb.
  jassert((bitWidth - effective_shift) < jfs_nr_bitvector_ty_bit_width);
  result |= value >> (bitWidth - effective_shift);
  result &= jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_rotate_right(const jfs_nr_bitvector_ty value,
                                        const jfs_nr_width_ty shift,
                                        const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  // ((_ rotate_right 0) t) stands for t
  // ((_ rotate_right i) t) abbreviates t if m = 1, and
  //   ((_ rotate_right |i-1|)
  //     (concat ((_ extract 0 0) t) ((_ extract |m-1| 1) t)))
  //   otherwise
  jfs_nr_bitvector_ty effective_shift = shift % bitWidth;
  // Shift bits to the right
  jassert(effective_shift < jfs_nr_bitvector_ty_bit_width);
  jfs_nr_bitvector_ty result = value >> effective_shift;
  // bitwise or with the bits that got shifted out and
  // should now appear in the msb.
  jassert((bitWidth - effective_shift) < jfs_nr_bitvector_ty_bit_width);
  result |= value << (bitWidth - effective_shift);
  result &= jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvand(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result = lhs & rhs;
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvor(const jfs_nr_bitvector_ty lhs,
                                const jfs_nr_bitvector_ty rhs,
                                const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result = lhs | rhs;
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvnand(const jfs_nr_bitvector_ty lhs,
                                  const jfs_nr_bitvector_ty rhs,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result =
      (~(lhs & rhs)) & jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvnor(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result =
      (~(lhs | rhs)) & jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvxor(const jfs_nr_bitvector_ty lhs,
                                 const jfs_nr_bitvector_ty rhs,
                                 const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result =
      (lhs ^ rhs) & jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvxnor(const jfs_nr_bitvector_ty lhs,
                                  const jfs_nr_bitvector_ty rhs,
                                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  jfs_nr_bitvector_ty result =
      (~(lhs ^ rhs)) & jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ jfs_nr_bitvector_ty jfs_nr_bvnot(const jfs_nr_bitvector_ty value,
                                 const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  jfs_nr_bitvector_ty result = (~value) & jfs_nr_get_bitvector_mask(bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

__device__ bool jfs_nr_bvult(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return lhs < rhs;
}

__device__ bool jfs_nr_bvule(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return lhs <= rhs;
}

__device__ bool jfs_nr_bvugt(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return lhs > rhs;
}

__device__ bool jfs_nr_bvuge(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  return lhs >= rhs;
}

__device__ bool jfs_nr_bvslt(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  // (bvslt s t) abbreviates:
  //  (or (and (= ((_ extract |m-1| |m-1|) s) #b1)
  //           (= ((_ extract |m-1| |m-1|) t) #b0))
  //      (and (= ((_ extract |m-1| |m-1|) s) ((_ extract |m-1| |m-1|) t))
  //           (bvult s t)))
  bool lhsNeg = lhs & jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  bool rhsNeg = rhs & jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  if (lhsNeg && !rhsNeg) {
    return true;
  }
  return (lhsNeg == rhsNeg) && jfs_nr_bvult(lhs, rhs, bitWidth);
}

__device__ bool jfs_nr_bvsle(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(lhs, bitWidth));
  jassert(jfs_nr_is_valid(rhs, bitWidth));
  // (bvsle s t) abbreviates:
  //  (or (and (= ((_ extract |m-1| |m-1|) s) #b1)
  //           (= ((_ extract |m-1| |m-1|) t) #b0))
  //      (and (= ((_ extract |m-1| |m-1|) s) ((_ extract |m-1| |m-1|) t))
  //           (bvule s t)))
  bool lhsNeg = lhs & jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  bool rhsNeg = rhs & jfs_nr_get_most_signficiant_bit_mask(bitWidth);
  if (lhsNeg && !rhsNeg) {
    return true;
  }
  return (lhsNeg == rhsNeg) && jfs_nr_bvule(lhs, rhs, bitWidth);
}

__device__ bool jfs_nr_bvsgt(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  // (bvsgt s t) abbreviates (bvslt t s)
  return jfs_nr_bvslt(rhs, lhs, bitWidth);
}

__device__ bool jfs_nr_bvsge(const jfs_nr_bitvector_ty lhs, const jfs_nr_bitvector_ty rhs,
                  const jfs_nr_width_ty bitWidth) {
  // (bvsge s t) abbreviates (bvsle t s)
  return jfs_nr_bvsle(rhs, lhs, bitWidth);
}

// FIXME: We should either handle byte aligned access specially, or provide
// another function for this.
// Convenience function for creating a BitVector
// from any arbitrary bit offset in a buffer. Offset
// is [lowbit, highbit].
__device__ jfs_nr_bitvector_ty jfs_nr_make_bitvector(const uint8_t* bufferData,
                                          const uint64_t bufferSize,
                                          const uint64_t lowBit,
                                          const uint64_t highBit) {
  jassert(highBit >= lowBit && "invalid lowBit and highBit");
  jassert(highBit < (bufferSize * 8));
  const uint64_t bitWidth = ((highBit - lowBit) + 1);
  const size_t lowBitByte = lowBit / 8;
  const size_t shiftOffset = lowBit % 8;
  // NOTE: doing `highBit / 8` to compute `highBitByte` is wrong. For [1,8]
  // that gives a highBit of 1 which is wrong for the loop below (should be 0).
  // const size_t highBitByte = (lowBitByte + ((BITWIDTH + 7) / 8)) - 1;
  const size_t highBitByte = (lowBitByte + (((highBit - lowBit) + 8) / 8)) - 1;
  jassert(lowBitByte < bufferSize);
  jassert(highBitByte < bufferSize);
  jfs_nr_bitvector_ty data = 0;
  uint8_t* dataView = reinterpret_cast<uint8_t*>(&data);
  jfs_nr_bitvector_ty dataMask = jfs_nr_get_bitvector_mask(bitWidth);
  // Copy byte-by-byte shifting if necessary
  for (size_t index = lowBitByte; index <= highBitByte; ++index) {
    const size_t viewIndex = index - lowBitByte;
    jassert(index < bufferSize);
    jassert(viewIndex < sizeof(data));
    uint8_t bufferByte = bufferData[index];
    dataView[viewIndex] |= (bufferByte >> shiftOffset);
    if (shiftOffset == 0) {
      // If there is no shift offset then we didn't shift any bits
      // out.
      continue;
    }
    // Doing the shift means we have zero bits in MSB rather than the actually
    // bits we want.
    uint8_t nextIterByteValue = 0;
    if ((index + 1) < bufferSize) {
      // Avoid out of bounds access
      nextIterByteValue = bufferData[index + 1];
    }
    dataView[viewIndex] |= (nextIterByteValue << (8 - shiftOffset));
  }
  // Now mask off the data
  data &= dataMask;
  return data;
}

__device__ void jfs_nr_write_bitvector(const jfs_nr_bitvector_ty bv,
                            const jfs_nr_width_ty bitWidth,
                            uint8_t* bufferData,
                            const uint64_t bufferSize,
                            const uint64_t bitOffset) {
  // Copy the bit vector so we can shift bits off as we go.
  // TODO: Or just mark it not const in the params...?
  // FIXME: Rework this algorithm to avoid the copy and shifting here if it
  // appears to be a bottleneck. (It seems okay since bit vectors are a small,
  // fixed size.)
  jfs_nr_bitvector_ty tempBv = bv;
  jassert(jfs_nr_is_valid(tempBv, bitWidth));
  const uint64_t endBit = bitOffset + bitWidth - 1;
  jassert(bufferData != nullptr);
  jassert(bitOffset < (bufferSize * 8));
  jassert(endBit < (bufferSize * 8));
  const size_t startByte = bitOffset / 8;
  const size_t endByte = endBit / 8;
  jassert(startByte < bufferSize);
  jassert(endByte < bufferSize);
  // Track the current bit offset over time as well as the bits left to write.
  uint64_t currentBitOffset = bitOffset;
  jfs_nr_width_ty bitsRemaining = bitWidth;
  // Track a view of the first byte of the bit vector.
  const uint8_t* bvByteView = reinterpret_cast<const uint8_t*>(&tempBv);
  // Write in the buffer byte by byte while preserving surrounding bits.
  for (size_t currentByte = startByte; currentByte <= endByte; ++currentByte) {
    // Example: `currentBitOffset` is 6. We can take bits [1, 0] from the BV and
    // write them to bits [7, 6] in buffer.
    const size_t offsetInByte = currentBitOffset % 8;
    size_t bitsToWrite = 8 - offsetInByte;
    // For the end byte, we have to cap `bitsToWrite` at the amount remaining.
    if (bitsToWrite > bitsRemaining) {
      bitsToWrite = bitsRemaining;
    }
    jassert(currentBitOffset < (bufferSize * 8));
    jassert(currentBitOffset + bitsToWrite - 1 <= endBit);
    const uint8_t bvMask = jfs_nr_get_bitvector_mask(bitsToWrite);
    const uint8_t bufferMask = ~(bvMask << offsetInByte);
    bufferData[currentByte] = (bufferData[currentByte] & bufferMask) |
                              ((*bvByteView & bvMask) << offsetInByte);
    // Shift the bits written off the bit vector.
    tempBv >>= bitsToWrite;
    currentBitOffset += bitsToWrite;
    bitsRemaining -= bitsToWrite;
  }
}

#ifdef __cplusplus
}
#endif
