#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                                     JFS
//
// Copyright 2017-2018 Daniel Liew
//
// This file is distributed under the MIT license.
// See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
#include "Messages.h"
#include <stdarg.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// Right now these are just a thin wrappers around fprintf.  In the future
// though we could log the messages to a file.

void jfs_info(const char* fmt, ...) {
  va_list args;
  va_start(args, fmt);
  fprintf(stderr, "JFS INFO: ");
  vfprintf(stderr, fmt, args);
  va_end(args);
}

void jfs_warning(const char* fmt, ...) {
  va_list args;
  va_start(args, fmt);
  fprintf(stderr, "JFS WARNING: ");
  vfprintf(stderr, fmt, args);
  va_end(args);
}

#ifdef __cplusplus
}
#endif
