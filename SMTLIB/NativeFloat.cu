#include "hip/hip_runtime.h"
//===----------------------------------------------------------------------===//
//
//                                     JFS
//
// Copyright 2017-2018 Daniel Liew
//
// This file is distributed under the MIT license.
// See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
// This is the implemenation of the runtime for SMTLIB Floats that
// uses native machine operations. It is written with a C compatible interface
// so that in the future we can easily use LLVM's JIT.
// 
// BDG: updated to run on CUDA-capable GPUs. Main changes:
//      * All functions marked with __device__
//      * CUDA doesn't have a persistent rounding mode state. Instead,
//        use specific floating point functions that specify the rounding
//        mode. Note that some functions still need to be updated (mostly
//        casts and "nearest int" functions) to support different modes

#include "SMTLIB/NativeFloat.h"
#include "SMTLIB/NativeBitVector.h"
#include "SMTLIB/jassert.h"
#include <fenv.h>
#include <math.h>
#include <string.h>

namespace {

// Generic version
template <typename T>
__device__ T jfs_nr_internal_make_float_from_buffer(const uint8_t* bufferData,
                                         const uint64_t bufferSize,
                                         const uint64_t lowBit) {
  // Just re-use bitvector method for now.
  // This assume little-endian which might things on other architectures.
  uint64_t highBit = lowBit + (sizeof(T) * 8) - 1;
  jassert((((highBit - lowBit) + 1) % 8) == 0 &&
          "Width requested should be whole bytes");
  jfs_nr_bitvector_ty rawBits =
      jfs_nr_make_bitvector(bufferData, bufferSize, lowBit, highBit);
  T result;
  // Copy into result
  memcpy(&result, &rawBits, sizeof(T));
  return result;
}

// Generic version
template <typename RetTy, typename ArgTy>
__device__ RetTy jfs_nr_internal_float_get_raw_bits(const ArgTy value) {
  static_assert(sizeof(RetTy) == sizeof(ArgTy), "Size mismatch");
  RetTy data = 0;
  memcpy(&data, &value, sizeof(RetTy));
  return data;
}
}

#ifdef __cplusplus
extern "C" {
#endif

__device__ uint32_t jfs_nr_float32_get_raw_bits(const jfs_nr_float32 value) {
  return jfs_nr_internal_float_get_raw_bits<uint32_t, jfs_nr_float32>(value);
}

__device__ uint64_t jfs_nr_float64_get_raw_bits(const jfs_nr_float64 value) {
  return jfs_nr_internal_float_get_raw_bits<uint64_t, jfs_nr_float64>(value);
}

__device__ jfs_nr_float32 jfs_nr_float32_get_infinity(bool positive) {
  if (positive)
    return INFINITY;
  return -INFINITY;
}

__device__ jfs_nr_float64 jfs_nr_float64_get_infinity(bool positive) {
  if (positive)
    return INFINITY;
  return -INFINITY;
}

__device__ jfs_nr_float32 jfs_nr_float32_get_zero(bool positive) {
  if (positive)
    return jfs_nr_bitcast_bv_to_float32(0x0);
  return jfs_nr_bitcast_bv_to_float32(UINT32_C(0x80000000));
}

__device__ jfs_nr_float64 jfs_nr_float64_get_zero(bool positive) {
  if (positive)
    return jfs_nr_bitcast_bv_to_float64(0x0);
  return jfs_nr_bitcast_bv_to_float64(UINT64_C(0x8000000000000000));
}

__device__ jfs_nr_float32 jfs_nr_float32_get_nan(bool quiet) {
  if (quiet)
    return jfs_nr_bitcast_bv_to_float32(UINT64_C(0x7fc00000));
  return jfs_nr_bitcast_bv_to_float32(UINT64_C(0x7f800001));
}

__device__ jfs_nr_float64 jfs_nr_float64_get_nan(bool quiet) {
  if (quiet)
    return jfs_nr_bitcast_bv_to_float64(UINT64_C(0x7ff8000000000000));
  return jfs_nr_bitcast_bv_to_float64(UINT64_C(0x7ff0000000000001));
}

__device__ bool jfs_nr_float32_is_normal(const jfs_nr_float32 value) {
  return isnormal(value) != 0;
}
__device__ bool jfs_nr_float64_is_normal(const jfs_nr_float64 value) {
  return isnormal(value) != 0;
}

__device__ bool jfs_nr_float32_is_subnormal(const jfs_nr_float32 value) {
  return fpclassify(value) == FP_SUBNORMAL;
}

__device__ bool jfs_nr_float64_is_subnormal(const jfs_nr_float64 value) {
  return fpclassify(value) == FP_SUBNORMAL;
}

__device__ bool jfs_nr_float32_is_zero(const jfs_nr_float32 value) {
  return fpclassify(value) == FP_ZERO;
}

__device__ bool jfs_nr_float64_is_zero(const jfs_nr_float64 value) {
  return fpclassify(value) == FP_ZERO;
}

__device__ bool jfs_nr_float32_is_infinite(const jfs_nr_float32 value) {
  return fpclassify(value) == FP_INFINITE;
}

__device__ bool jfs_nr_float64_is_infinite(const jfs_nr_float64 value) {
  return fpclassify(value) == FP_INFINITE;
}

__device__ bool jfs_nr_float32_is_positive(const jfs_nr_float32 value) {
  // +zero is treated as positive
  if (jfs_nr_float32_get_raw_bits(value) == UINT64_C(0x0))
    return true;
  return jfs_nr_float32_get_zero(/*positive=*/true) < value;
}

__device__ bool jfs_nr_float64_is_positive(const jfs_nr_float64 value) {
  // +zero is treated as positive
  if (jfs_nr_float64_get_raw_bits(value) == UINT64_C(0x0))
    return true;
  return jfs_nr_float64_get_zero(/*positive=*/true) < value;
}

__device__ bool jfs_nr_float32_is_negative(const jfs_nr_float32 value) {
  // -zero is treated as negative
  if (jfs_nr_float32_get_raw_bits(value) == UINT32_C(0x80000000))
    return true;
  return value < jfs_nr_float32_get_zero(/*positive=*/false);
}

__device__ bool jfs_nr_float64_is_negative(const jfs_nr_float64 value) {
  // -zero is treated as negative
  if (jfs_nr_float64_get_raw_bits(value) == UINT64_C(0x8000000000000000))
    return true;
  return value < jfs_nr_float64_get_zero(/*positive=*/false);
}

__device__ bool jfs_nr_float32_is_nan(const jfs_nr_float32 value) { return isnan(value); }

__device__ bool jfs_nr_float64_is_nan(const jfs_nr_float64 value) { return isnan(value); }

__device__ bool jfs_nr_float32_ieee_equals(const jfs_nr_float32 lhs,
                                const jfs_nr_float32 rhs) {
  return lhs == rhs;
}

__device__ bool jfs_nr_float64_ieee_equals(const jfs_nr_float64 lhs,
                                const jfs_nr_float64 rhs) {
  return lhs == rhs;
}

__device__ bool jfs_nr_float32_smtlib_equals(const jfs_nr_float32 lhs,
                                  const jfs_nr_float32 rhs) {
  // In SMT-LIBv2 no distinction is made between the different types of NaN
  /*
   *  (set-logic QF_FPBV)
      (declare-const x (_ BitVec 32))
      (declare-const y (_ BitVec 32))
      (assert (not (= x y)))
      (assert (fp.isNaN ((_ to_fp 8 24) x)))
      (assert (fp.isNaN ((_ to_fp 8 24) y)))
      (assert
        (not
          (=
            ((_ to_fp 8 24) x)
            ((_ to_fp 8 24) y)
          )
        )
      )
      (check-sat)
      unsat
  */
  bool lhsIsNaN = isnan(lhs);
  bool rhsIsNaN = isnan(rhs);
  if (lhsIsNaN && rhsIsNaN) {
    return true;
  }
  // Positive and negative 0 are distinct but C's `==` operator considers them
  // equal so just do bit comparison.
  return jfs_nr_float32_get_raw_bits(lhs) == jfs_nr_float32_get_raw_bits(rhs);
}

__device__ bool jfs_nr_float64_smtlib_equals(const jfs_nr_float64 lhs,
                                  const jfs_nr_float64 rhs) {
  // In SMT-LIBv2 no distinction is made between the different types of NaN
  bool lhsIsNaN = isnan(lhs);
  bool rhsIsNaN = isnan(rhs);
  if (lhsIsNaN && rhsIsNaN) {
    return true;
  }
  // Positive and negative 0 are distinct but C's `==` operator considers them
  // equal so just do bit comparison.
  return jfs_nr_float64_get_raw_bits(lhs) == jfs_nr_float64_get_raw_bits(rhs);
}

__device__ jfs_nr_float32 jfs_nr_float32_abs(const jfs_nr_float32 value) {
  return ::fabsf(value);
}

__device__ jfs_nr_float64 jfs_nr_float64_abs(const jfs_nr_float64 value) {
  return ::fabs(value);
}

__device__ jfs_nr_float32 jfs_nr_float32_neg(const jfs_nr_float32 value) {
  uint32_t rawBits = jfs_nr_float32_get_raw_bits(value);
  // Flip sign bit
  rawBits ^= UINT32_C(0x80000000);
  return jfs_nr_bitcast_bv_to_float32(rawBits);
}

// FIXME: We are assuming that FE_TONEAREST is RNE but it could be
// RNA which would be wrong. We should build a target specific version
#ifndef __x86_64
#error FIXME UNSUPPORTED PLATFORM
#endif
#define fesetround(x) 0
// FIXME: Do we need a barrier here to prevent Clang moving around the calls?
#define JFS_NR_SET_RM(X)                                                       \
  {                                                                            \
    switch (X) {                                                               \
    case JFS_RM_RNE: {                                                         \
      /* This is the default so do nothing */                                  \
      break;                                                                   \
    }                                                                          \
    case JFS_RM_RNA: {                                                         \
      /* FIXME: Not supported by C */                                          \
      JFS_RUNTIME_FAIL()                                                       \
      break;                                                                   \
    }                                                                          \
    case JFS_RM_RTP: {                                                         \
      break;                                                                   \
    }                                                                          \
    case JFS_RM_RTN: {                                                         \
      break;                                                                   \
    }                                                                          \
    case JFS_RM_RTZ: {                                                         \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      JFS_RUNTIME_FAIL()                                                       \
    }                                                                          \
  }
#define JFS_NR_RESET_RM(PREVIOUS)                                              \
  {                                                                            \
    switch (PREVIOUS) {                                                        \
    case JFS_RM_RNE: {                                                         \
      /* The rounding mode switched was the default so do nothing */           \
      break;                                                                   \
    }                                                                          \
    case JFS_RM_RTP:                                                           \
    case JFS_RM_RTN:                                                           \
    case JFS_RM_RTZ: {                                                         \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      JFS_RUNTIME_FAIL();                                                      \
    }                                                                          \
  }

// FIXME: This is a hack. Clang doesn't support the FENV_ACCESS pragma
// and so incorrectly optimizes the code so the rounding mode change and
// floating point operation get re-ordered. Using `asm volatile ("" : : :
// "memory")`
// didn't seem to work unfortunately so for now put this attribute on all
// functions
// that might change the rounding mode.
// See https://bugs.llvm.org/show_bug.cgi?id=8100
#define NO_OPT 

__device__ NO_OPT jfs_nr_float32 jfs_nr_float32_add(JFS_NR_RM rm, const jfs_nr_float32 lhs,
                                         const jfs_nr_float32 rhs) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fadd_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __fadd_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __fadd_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __fadd_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_float64_add(JFS_NR_RM rm, const jfs_nr_float64 lhs,
                                         const jfs_nr_float64 rhs) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __dadd_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __dadd_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __dadd_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __dadd_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float32 jfs_nr_float32_sub(JFS_NR_RM rm, const jfs_nr_float32 lhs,
                                         const jfs_nr_float32 rhs) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fadd_rn(lhs,-rhs) ; break;
    case JFS_RM_RTP: result = __fadd_ru(lhs,-rhs) ; break;
    case JFS_RM_RTN: result = __fadd_rd(lhs,-rhs) ; break;
    case JFS_RM_RTZ: result = __fadd_rz(lhs,-rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_float64_sub(JFS_NR_RM rm, const jfs_nr_float64 lhs,
                                         const jfs_nr_float64 rhs) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __dadd_rn(lhs,-rhs) ; break;
    case JFS_RM_RTP: result = __dadd_ru(lhs,-rhs) ; break;
    case JFS_RM_RTN: result = __dadd_rd(lhs,-rhs) ; break;
    case JFS_RM_RTZ: result = __dadd_rz(lhs,-rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float32 jfs_nr_float32_mul(JFS_NR_RM rm, const jfs_nr_float32 lhs,
                                         const jfs_nr_float32 rhs) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fmul_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __fmul_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __fmul_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __fmul_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_float64_mul(JFS_NR_RM rm, const jfs_nr_float64 lhs,
                                         const jfs_nr_float64 rhs) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __dmul_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __dmul_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __dmul_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __dmul_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

// Use the attribute to prevent UBSan from raising errors when doing floating
// point division by zero.
#define ALLOW_DIV_BY_ZERO __attribute__((no_sanitize("float-divide-by-zero")))
__device__ NO_OPT ALLOW_DIV_BY_ZERO jfs_nr_float32 jfs_nr_float32_div(
    JFS_NR_RM rm, const jfs_nr_float32 lhs, const jfs_nr_float32 rhs) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fdiv_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __fdiv_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __fdiv_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __fdiv_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT ALLOW_DIV_BY_ZERO jfs_nr_float64 jfs_nr_float64_div(
    JFS_NR_RM rm, const jfs_nr_float64 lhs, const jfs_nr_float64 rhs) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __ddiv_rn(lhs,rhs) ; break;
    case JFS_RM_RTP: result = __ddiv_ru(lhs,rhs) ; break;
    case JFS_RM_RTN: result = __ddiv_rd(lhs,rhs) ; break;
    case JFS_RM_RTZ: result = __ddiv_rz(lhs,rhs) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}
#undef ALLOW_DIV_BY_ZERO

__device__ NO_OPT jfs_nr_float32 jfs_nr_float32_fma(JFS_NR_RM rm, const jfs_nr_float32 a,
                                         const jfs_nr_float32 b,
                                         const jfs_nr_float32 c) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fmaf_rn(a,b,c) ; break;
    case JFS_RM_RTP: result = __fmaf_ru(a,b,c) ; break;
    case JFS_RM_RTN: result = __fmaf_rd(a,b,c) ; break;
    case JFS_RM_RTZ: result = __fmaf_rz(a,b,c) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_float64_fma(JFS_NR_RM rm, const jfs_nr_float64 a,
                                         const jfs_nr_float64 b,
                                         const jfs_nr_float64 c) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fma_rn(a,b,c) ; break;
    case JFS_RM_RTP: result = __fma_ru(a,b,c) ; break;
    case JFS_RM_RTN: result = __fma_rd(a,b,c) ; break;
    case JFS_RM_RTZ: result = __fma_rz(a,b,c) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float32 jfs_nr_float32_sqrt(JFS_NR_RM rm,
                                          const jfs_nr_float32 value) {
  jfs_nr_float32 result;
  switch (rm) {
    case JFS_RM_RNE: result = __fsqrt_rn(value) ; break;
    case JFS_RM_RTP: result = __fsqrt_ru(value) ; break;
    case JFS_RM_RTN: result = __fsqrt_rd(value) ; break;
    case JFS_RM_RTZ: result = __fsqrt_rz(value) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_float64_sqrt(JFS_NR_RM rm,
                                          const jfs_nr_float64 value) {
  jfs_nr_float64 result;
  switch (rm) {
    case JFS_RM_RNE: result = __dsqrt_rn(value) ; break;
    case JFS_RM_RTP: result = __dsqrt_ru(value) ; break;
    case JFS_RM_RTN: result = __dsqrt_rd(value) ; break;
    case JFS_RM_RTZ: result = __dsqrt_rz(value) ; break;
    case JFS_RM_RNA: JFS_RUNTIME_FAIL() ; break;
    default: JFS_RUNTIME_FAIL();
  }
  return result;
}

__device__ NO_OPT jfs_nr_float32
jfs_nr_float32_round_to_integral(JFS_NR_RM rm, const jfs_nr_float32 value) {
  // FIXME: We can use the `round()` C math library function to support
  // JFS_RM_RNA
  JFS_NR_SET_RM(rm)
  jfs_nr_float32 result = nearbyintf(value);
  JFS_NR_RESET_RM(rm)
  return result;
}

__device__ NO_OPT jfs_nr_float64
jfs_nr_float64_round_to_integral(JFS_NR_RM rm, const jfs_nr_float64 value) {
  // FIXME: We can use the `round()` C math library function to support
  // JFS_RM_RNA
  JFS_NR_SET_RM(rm)
  jfs_nr_float64 result = nearbyint(value);
  JFS_NR_RESET_RM(rm)
  return result;
}

// Use this attribute to prevent UBSan from raising errors when doing float
// casts.
#define ALLOW_OVERFLOW __attribute__((no_sanitize("float-cast-overflow")))

__device__ NO_OPT ALLOW_OVERFLOW jfs_nr_float32
jfs_nr_convert_float64_to_float32(JFS_NR_RM rm, const jfs_nr_float64 value) {
  JFS_NR_SET_RM(rm)
  jfs_nr_float32 result = (jfs_nr_float32)value;
  JFS_NR_RESET_RM(rm)
  return result;
}

__device__ jfs_nr_float64 jfs_nr_convert_float32_to_float64(const jfs_nr_float32 value) {
  // No rounding mode is required because every jfs_nr_float32 value can be
  // precisely represented as a jfs_nr_float64.
  return (jfs_nr_float64)value;
}

__device__ NO_OPT jfs_nr_float32 jfs_nr_convert_from_unsigned_bv_to_float32(
    JFS_NR_RM rm, const jfs_nr_bitvector_ty value,
    const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  JFS_NR_SET_RM(rm)
  jfs_nr_float32 result = (jfs_nr_float32)value;
  JFS_NR_RESET_RM(rm)
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_convert_from_unsigned_bv_to_float64(
    JFS_NR_RM rm, const jfs_nr_bitvector_ty value,
    const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  JFS_NR_SET_RM(rm)
  jfs_nr_float64 result = (jfs_nr_float64)value;
  JFS_NR_RESET_RM(rm)
  return result;
}

__device__ NO_OPT jfs_nr_float32 jfs_nr_convert_from_signed_bv_to_float32(
    JFS_NR_RM rm, const jfs_nr_bitvector_ty value,
    const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  // If the bitvector is negative turn it into a positive value and negate
  // the float afterwards.
  bool shouldNegateFloat = false;
  jfs_nr_bitvector_ty positiveBv = value;
  if (jfs_nr_bvslt(positiveBv, 0, bitWidth)) {
    shouldNegateFloat = true;
    positiveBv = jfs_nr_bvneg(value, bitWidth);
  }
  jassert(jfs_nr_bvsge(positiveBv, 0, bitWidth));
  JFS_NR_SET_RM(rm)
  jfs_nr_float32 result = (jfs_nr_float32)positiveBv;
  JFS_NR_RESET_RM(rm)
  if (shouldNegateFloat) {
    result = jfs_nr_float32_neg(result);
  }
  return result;
}

__device__ NO_OPT jfs_nr_float64 jfs_nr_convert_from_signed_bv_to_float64(
    JFS_NR_RM rm, const jfs_nr_bitvector_ty value,
    const jfs_nr_width_ty bitWidth) {
  jassert(jfs_nr_is_valid(value, bitWidth));
  // If the bitvector is negative turn it into a positive value and negate
  // the float afterwards.
  bool shouldNegateFloat = false;
  jfs_nr_bitvector_ty positiveBv = value;
  if (jfs_nr_bvslt(positiveBv, 0, bitWidth)) {
    shouldNegateFloat = true;
    positiveBv = jfs_nr_bvneg(value, bitWidth);
  }
  jassert(jfs_nr_bvsge(positiveBv, 0, bitWidth));
  JFS_NR_SET_RM(rm)
  jfs_nr_float64 result = (jfs_nr_float64)positiveBv;
  JFS_NR_RESET_RM(rm)
  if (shouldNegateFloat) {
    result = jfs_nr_float64_neg(result);
  }
  return result;
}

// FIXME: We should probably pick a single value to represent the
// undefined case so we can test for it.
__device__ NO_OPT ALLOW_OVERFLOW jfs_nr_bitvector_ty jfs_nr_float32_convert_to_unsigned_bv(
    JFS_NR_RM rm, jfs_nr_float32 value, const jfs_nr_width_ty bitWidth) {
  JFS_NR_SET_RM(rm)
  jfs_nr_bitvector_ty result = (jfs_nr_bitvector_ty)value;
  JFS_NR_RESET_RM(rm)
  // Mask off result
  result = jfs_nr_get_bitvector_mod(result, bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

// FIXME: We should probably pick a single value to represent the
// undefined case so we can test for it.
__device__ NO_OPT ALLOW_OVERFLOW jfs_nr_bitvector_ty jfs_nr_float64_convert_to_unsigned_bv(
    JFS_NR_RM rm, jfs_nr_float64 value, const jfs_nr_width_ty bitWidth) {
  JFS_NR_SET_RM(rm)
  jfs_nr_bitvector_ty result = (jfs_nr_bitvector_ty)value;
  JFS_NR_RESET_RM(rm)
  // Mask off result
  result = jfs_nr_get_bitvector_mod(result, bitWidth);
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

// FIXME: We should probably pick a single value to represent the
// undefined case so we can test for it.
__device__ NO_OPT ALLOW_OVERFLOW jfs_nr_bitvector_ty jfs_nr_float32_convert_to_signed_bv(
    JFS_NR_RM rm, jfs_nr_float32 value, const jfs_nr_width_ty bitWidth) {
  bool shouldNegateResult = false;
  jfs_nr_float32 positiveFloat = value;
  // if float is negative turn it into a positive value and negate the
  // bitvector afterwards.
  if (jfs_nr_float32_is_negative(value)) {
    positiveFloat = jfs_nr_float32_neg(value);
    shouldNegateResult = true;
  }
  jassert(jfs_nr_float32_is_positive(positiveFloat));
  JFS_NR_SET_RM(rm)
  jfs_nr_bitvector_ty result = (jfs_nr_bitvector_ty)positiveFloat;
  JFS_NR_RESET_RM(rm)
  // Mask off result
  result = jfs_nr_get_bitvector_mod(result, bitWidth);
  if (shouldNegateResult) {
    result = jfs_nr_bvneg(result, bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

// FIXME: We should probably pick a single value to represent the
// undefined case so we can test for it.
__device__ NO_OPT ALLOW_OVERFLOW jfs_nr_bitvector_ty jfs_nr_float64_convert_to_signed_bv(
    JFS_NR_RM rm, jfs_nr_float64 value, const jfs_nr_width_ty bitWidth) {
  bool shouldNegateResult = false;
  jfs_nr_float64 positiveFloat = value;
  // if float is negative turn it into a positive value and negate the
  // bitvector afterwards.
  if (jfs_nr_float64_is_negative(value)) {
    positiveFloat = jfs_nr_float64_neg(value);
    shouldNegateResult = true;
  }
  jassert(jfs_nr_float64_is_positive(positiveFloat));
  JFS_NR_SET_RM(rm)
  jfs_nr_bitvector_ty result = (jfs_nr_bitvector_ty)positiveFloat;
  JFS_NR_RESET_RM(rm)
  // Mask off result
  result = jfs_nr_get_bitvector_mod(result, bitWidth);
  if (shouldNegateResult) {
    result = jfs_nr_bvneg(result, bitWidth);
  }
  jassert(jfs_nr_is_valid(result, bitWidth));
  return result;
}

#undef JFS_NR_SET_RM
#undef JFS_NR_RESET_RM
#undef ALLOW_OVERFLOW

__device__ jfs_nr_float32 jfs_nr_float32_rem(const jfs_nr_float32 lhs,
                                  const jfs_nr_float32 rhs) {
  return remainderf(lhs, rhs);
}

__device__ jfs_nr_float64 jfs_nr_float64_rem(const jfs_nr_float64 lhs,
                                  const jfs_nr_float64 rhs) {
  return remainder(lhs, rhs);
}

__device__ jfs_nr_float64 jfs_nr_float64_neg(const jfs_nr_float64 value) {
  uint64_t rawBits = jfs_nr_float64_get_raw_bits(value);
  // Flip sign bit
  rawBits ^= UINT64_C(0x8000000000000000);
  return jfs_nr_bitcast_bv_to_float64(rawBits);
}

__device__ jfs_nr_float32 jfs_nr_float32_min(const jfs_nr_float32 lhs,
                                  const jfs_nr_float32 rhs) {
  return ::fminf(lhs, rhs);
}

__device__ jfs_nr_float64 jfs_nr_float64_min(const jfs_nr_float64 lhs,
                                  const jfs_nr_float64 rhs) {
  return ::fmin(lhs, rhs);
}

__device__ jfs_nr_float32 jfs_nr_float32_max(const jfs_nr_float32 lhs,
                                  const jfs_nr_float32 rhs) {
  return ::fmaxf(lhs, rhs);
}

__device__ jfs_nr_float64 jfs_nr_float64_max(const jfs_nr_float64 lhs,
                                  const jfs_nr_float64 rhs) {
  return ::fmax(lhs, rhs);
}

__device__ bool jfs_nr_float32_leq(const jfs_nr_float32 lhs, const jfs_nr_float32 rhs) {
  return lhs <= rhs;
}

__device__ bool jfs_nr_float64_leq(const jfs_nr_float64 lhs, const jfs_nr_float64 rhs) {
  return lhs <= rhs;
}

__device__ bool jfs_nr_float32_lt(const jfs_nr_float32 lhs, const jfs_nr_float32 rhs) {
  return lhs < rhs;
}

__device__ bool jfs_nr_float64_lt(const jfs_nr_float64 lhs, const jfs_nr_float64 rhs) {
  return lhs < rhs;
}

__device__ bool jfs_nr_float32_gt(const jfs_nr_float32 lhs, const jfs_nr_float32 rhs) {
  return lhs > rhs;
}

__device__ bool jfs_nr_float64_gt(const jfs_nr_float64 lhs, const jfs_nr_float64 rhs) {
  return lhs > rhs;
}

__device__ bool jfs_nr_float32_geq(const jfs_nr_float32 lhs, const jfs_nr_float32 rhs) {
  return lhs >= rhs;
}

__device__ bool jfs_nr_float64_geq(const jfs_nr_float64 lhs, const jfs_nr_float64 rhs) {
  return lhs >= rhs;
}

__device__ jfs_nr_float32 jfs_nr_bitcast_bv_to_float32(const jfs_nr_bitvector_ty value) {
  jassert((value & UINT64_C(0xffffffff00000000)) == 0);
  jfs_nr_float32 data = 0.0;
  memcpy(&data, &value, sizeof(data));
  return data;
}

__device__ jfs_nr_float64 jfs_nr_bitcast_bv_to_float64(const jfs_nr_bitvector_ty value) {
  jfs_nr_float64 data = 0.0;
  memcpy(&data, &value, sizeof(data));
  return data;
}

// Note significand does not contain implicit bit
__device__ jfs_nr_float32
jfs_nr_make_float32_from_triple(const jfs_nr_bitvector_ty sign,
                                const jfs_nr_bitvector_ty exponent,
                                const jfs_nr_bitvector_ty significand) {
  static_assert((sizeof(jfs_nr_bitvector_ty) * 8) >= 32, "not enough bits");
  jassert((sign & (~(UINT64_C(0x1)))) == 0);             // only 1 bit
  jassert((exponent & (~(UINT64_C(0xff)))) == 0);        // only 8-bits
  jassert((significand & (~(UINT64_C(0x7fffff)))) == 0); // only 23-bits
  jfs_nr_bitvector_ty rawBits = significand;
  rawBits |= (exponent << 23);
  rawBits |= (sign << 31);
  return jfs_nr_bitcast_bv_to_float32(rawBits);
}

__device__ jfs_nr_float64
jfs_nr_make_float64_from_triple(const jfs_nr_bitvector_ty sign,
                                const jfs_nr_bitvector_ty exponent,
                                const jfs_nr_bitvector_ty significand) {
  // TODO: Finish!
  static_assert((sizeof(jfs_nr_bitvector_ty) * 8) >= 64, "not enough bits");
  jassert((sign & (~(UINT64_C(0x1)))) == 0);       // only 1 bit
  jassert((exponent & (~(UINT64_C(0x7ff)))) == 0); // only 11-bits
  jassert((significand & (~(UINT64_C(0x000fffffffffffff)))) ==
          0); // only 52-bits
  jfs_nr_bitvector_ty rawBits = significand;
  rawBits |= (exponent << 52);
  rawBits |= (sign << 63);
  return jfs_nr_bitcast_bv_to_float64(rawBits);
}

__device__ jfs_nr_float32 jfs_nr_make_float32_from_buffer(const uint8_t* bufferData,
                                               const uint64_t bufferSize,
                                               const uint64_t lowBit) {
  return jfs_nr_internal_make_float_from_buffer<jfs_nr_float32>(
      bufferData, bufferSize, lowBit);
}

__device__ jfs_nr_float64 jfs_nr_make_float64_from_buffer(const uint8_t* bufferData,
                                               const uint64_t bufferSize,
                                               const uint64_t lowBit) {
  return jfs_nr_internal_make_float_from_buffer<jfs_nr_float64>(
      bufferData, bufferSize, lowBit);
}

#ifdef __cplusplus
}
#endif
