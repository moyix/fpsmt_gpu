//===----------------------------------------------------------------------===//
//
//                                     JFS
//
// Copyright 2017-2018 Daniel Liew
//
// This file is distributed under the MIT license.
// See LICENSE.txt for details.
//
//===----------------------------------------------------------------------===//
// FIXME: Make the layout consistent between srcdir and build directory
#include "SMTLIB/Core.h"
#include "jassert.h"

// FIXME: Should we specialise for byte aligned access?
__device__ bool makeBoolFrom(BufferRef<const uint8_t> buffer, const uint64_t lowBit,
                  const uint64_t highBit) {
  jassert(highBit >= lowBit && "invalid lowBit and highBit");
  const size_t bitWidth = (highBit - lowBit) + 1;
  jassert(bitWidth <= 8 && "Too many bits");
  const size_t lowBitByte = lowBit / 8;
  const size_t highBitByte = highBit / 8;
  jassert(lowBitByte < buffer.getSize());
  jassert(highBitByte < buffer.getSize());
  uint8_t data = 0;
  const size_t shiftOffset = lowBit % 8;
  uint8_t dataMask = 0;
  if (bitWidth < 8) {
    dataMask = (UINT8_C(1) << bitWidth) - 1;
  } else {
    dataMask = UINT8_MAX;
  }
  // Read from firstByte
  uint8_t bufferByte = buffer.get()[lowBitByte];
  data = (bufferByte >> shiftOffset);
  // If necessary read bits from the subsequent byte if we
  // are stradling bytes
  if (highBitByte > lowBitByte) {
    jassert(shiftOffset > 0);
    jassert(highBitByte < buffer.getSize());
    uint8_t nextBufferByte = buffer.get()[highBitByte];
    data |= (nextBufferByte << (8 - shiftOffset));
  }
  data &= dataMask;
  if (data == 0)
    return false;
  return true;
}
